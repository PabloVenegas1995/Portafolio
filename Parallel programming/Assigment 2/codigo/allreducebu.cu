#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


using namespace std;

__global__ void allreduce(int *in, int *out) {
	int id = blockDim.x*blockIdx.x + threadIdx.x;
	int thid = threadIdx.x;
	int bdim = blockDim.x;
	int bid = blockIdx.x;

	extern __shared__ int sharray[];
	sharray[thid] = in[id]; 

	// todas las hebras en su bloque escriben en mem compartida
	// luego hay que esperar que todas terminn
	__syncthreads();

	//for(int i=bdim/2; i > 0; i >>= 1){
	for(int i=bdim/2; i > 0; i/=2){
		//printf(" i %d thid %d bid %d\n", i, thid, bid);
		if(thid < i){
			sharray[thid] += sharray[thid + i];
		}
		__syncthreads();
	}

	if(thid == 0){
		out[bid] = sharray[0];
	}

}

void initA(int *in, int N){
        for(int i=0; i<N; i++)
                in[i] = i;
}

void print(int *in, int N){
        for(int i=0; i<N; i++)
                printf("%d ", in[i]);
        printf("\n");
}


int main(int argc, char *argv[]){
	
	if(argc != 3){
		cout<<" USO "<<argv[0]<<" N K (Threads/Block)\n";
		return 1;
	}

	int N = atoi(argv[1]);
        int K = atoi(argv[2]);
	int midev;
        hipGetDevice(&midev);

	cout<<" N "<<N<<" K "<<K<<endl;
        int nb = (N+1)/K;
	cout<<" K "<<K<<" nb "<<nb<<" nb * sizeof(int) "<<nb*sizeof(int)<<endl;

        int size = N*sizeof(int);
        int *in = (int *)malloc(size);
        int *oner = (int *)malloc(sizeof(int));

        initA(in,N);
        print(in,N);

	int *d_in, *d_out, *d_one;
	hipMalloc(&d_in, size);
	hipMalloc(&d_out, nb*sizeof(int));
	hipMalloc(&d_one, sizeof(int));
	hipMemcpy(d_in, in, size, hipMemcpyHostToDevice);
	
        int tpb = K;
        int sharebytes = K*sizeof(int);
	//TIMERSTART(allreduce)
	allreduce<<<nb,tpb,sharebytes>>>(d_in,d_out); //blocks = N/K  

        hipError_t err = hipGetLastError();
        if (err != hipSuccess) 
                std::cout<<"Error: "<<hipGetErrorString(err)<<std::endl;

	tpb = nb;

	allreduce<<<1,tpb,sharebytes>>>(d_out,d_one); //blocks = N/K  
	//TIMERSTOP(allreduce)

	hipMemcpy(oner, d_one, sizeof(int), hipMemcpyDeviceToHost);

	hipFree(d_in);
	hipFree(d_out);
	hipFree(d_one);

	cout<<" out\n";
	cout<<" oner "<<*oner<<endl;

	free(in);
	free(oner);

	cout<<" fin \n";
}
