#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include "CImg.h"
#include <stdio.h>
#include <iostream>

using namespace std;
using namespace cimg_library;

/*__global__ void rgbsame(unsigned char * d_src, unsigned char * d_dst, int width, int height)
{
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    if (x >= width || y >= height)
        return;

    unsigned char r = d_src[y * width + x];
    unsigned char g = d_src[(height + y ) * width + x];
    unsigned char b = d_src[(height * 2 + y) * width + x];

    d_dst[y * width + x] = r;
    d_dst[(height + y)*width +x] = g;
    d_dst[(height * 2 + y)*width +x] = b;

}*/

__global__ void gray(unsigned char *d_src, unsigned char * d_dst, int width, int height, int proporcion){
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;


//int height_d = height/2, width_d = width/2;
    if (x >= width || y >= height)return;
if(x%2 == 0 && y%2 != 0){
    unsigned char r = d_src[y * width + x];
    unsigned char g = d_src[(height + y ) * width + x];
    unsigned char b = d_src[(height * 2 + y) * width + x];

    d_dst[(y/proporcion * width/proporcion + x/proporcion)] = r;
    d_dst[((height/proporcion + y/proporcion)*width/proporcion +x/proporcion)] = g;
    d_dst[((height*2/proporcion + y/proporcion)*width/proporcion +x/proporcion)] = b;
	}
}

int main(int argc, char *argv[]) {

    if(argc != 2){
        cout<<"uso: "<<argv[0]<<" image "<<endl;
    }

    CImg<unsigned char> src(argv[1]);
    string out = string(argv[1]) + ".gray";
	
    int proporcion = atoi(argv[2]);

    int width = src.width();
    int height = src.height();

	cout << " width " << width << " height " << height << endl;

    unsigned long size = src.size();
    cout<<" src size "<<size<<endl;

    unsigned char *h_src = src.data();

    CImg<unsigned char> dst(width/proporcion, height/proporcion, 1, 3);
    unsigned char *h_dst = dst.data();
    cout<<" dst size "<<dst.size()<<endl;

    unsigned char *d_src;
    unsigned char *d_dst;

    hipMalloc((void**)&d_src, size);
    hipMalloc((void**)&d_dst, size/(proporcion*proporcion)*sizeof(unsigned char));

    hipMemcpy(d_src, h_src, size, hipMemcpyHostToDevice);

    dim3 blkDim (16, 16, 1);
    dim3 grdDim ((width + 15)/16, (height + 15)/16, 1);

    gray<<<grdDim, blkDim>>>(d_src, d_dst, width, height, proporcion);

    hipMemcpy(h_dst, d_dst, size/(proporcion*proporcion), hipMemcpyDeviceToHost);

    hipFree(d_src);
    hipFree(d_dst);
    dst.save(out.c_str());
    dst.display();
 

    return 0;
}
